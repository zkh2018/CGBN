#include "hip/hip_runtime.h"
#include "msm/asm_cuda.h"

#include <gmp.h>
#include "cgbn/cgbn.h"
#include "cgbn_fp.h"
#include "cgbn_alt_bn128_g1.cuh"
#include <stdint.h>

using namespace gpu;

__global__ void kernel_msm_mul(
    uint64_t *a, uint64_t *b, uint64_t *c){
    limb_t r[12];

    asm(
      "{\n\t"
      ".reg .u64 c;\n\t"
      ".reg .u64 nc;\n\t"
      ".reg .u64 t;\n\t"
      //r[0], c = a[0] * b[0] 
      "mad.lo.cc.u64 %0, %12, %18, 0;\n\t"
      "madc.hi.cc.u64 c, %12, %18, 0;\n\t"
      
      //r[1], c = a[0] * b[1] + c
      "madc.lo.cc.u64 %1, %12, %19, c;\n\t"
      "madc.hi.cc.u64 c, %12, %19, 0;\n\t"
    
      //r[2], c = a[0] * b[2] + c
      "madc.lo.cc.u64 %2, %12, %20, c;\n\t"
      "madc.hi.cc.u64 c, %12, %20, 0;\n\t"

      //r[3], c = a[0] * b[3] + c
      "madc.lo.cc.u64 %3, %12, %21, c;\n\t"
      "madc.hi.cc.u64 c, %12, %21, 0;\n\t"

      //r[4], c = a[0] * b[4] + c
      "madc.lo.cc.u64 %4, %12, %22, c;\n\t"
      "madc.hi.cc.u64 c, %12, %22, 0;\n\t"

      //r[5], c = a[0] * b[5] + c
      "madc.lo.cc.u64 %5, %12, %23, c;\n\t"
      "madc.hi.u64 %6, %12, %23, 0;\n\t"


      //r[1], c = a[1] * b[0] + c
      "mad.lo.cc.u64 %1, %13, %18, %1;\n\t"
      "madc.hi.cc.u64 c, %13, %18, 0;\n\t"
      
      //t = r[2] + c
      "addc.cc.u64 t, %2, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      //r[2], c = a[1] * b[1] + c
      "mad.lo.cc.u64 %2, %13, %19, t;\n\t"
      "madc.hi.cc.u64 c, %13, %19, nc;\n\t"

      "addc.cc.u64 t, %3, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      //r[3], c = a[1] * b[2] + c
      "mad.lo.cc.u64 %3, %13, %20, t;\n\t"
      "madc.hi.cc.u64 c, %13, %20, nc;\n\t"

      "addc.cc.u64 t, %4, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      //r[4], c = a[1] * b[3] + c
      "mad.lo.cc.u64 %4, %13, %21, t;\n\t"
      "madc.hi.cc.u64 c, %13, %21, nc;\n\t"

      "addc.cc.u64 t, %5, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      //r[5], c = a[1] * b[4] + c
      "mad.lo.cc.u64 %5, %13, %22, t;\n\t"
      "madc.hi.cc.u64 c, %13, %22, nc;\n\t"

      "addc.cc.u64 t, %6, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      //r[6], c = a[1] * b[5] + c
      "mad.lo.cc.u64 %6, %13, %23, t;\n\t"
      "madc.hi.u64 %7, %13, %23, nc;\n\t"


      //r[2], c = a[2] * b[0] + c
      "mad.lo.cc.u64 %2, %14, %18, %2;\n\t"
      "madc.hi.cc.u64 c, %14, %18, 0;\n\t"
      
      "addc.cc.u64 t, %3, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %3, %14, %19, t;\n\t"
      "madc.hi.cc.u64 c, %14, %19, nc;\n\t"
      
      "addc.cc.u64 t, %4, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %4, %14, %20, t;\n\t"
      "madc.hi.cc.u64 c, %14, %20, nc;\n\t"

      "addc.cc.u64 t, %5, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %5, %14, %21, t;\n\t"
      "madc.hi.cc.u64 c, %14, %21, nc;\n\t"

      "addc.cc.u64 t, %6, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %6, %14, %22, t;\n\t"
      "madc.hi.cc.u64 c, %14, %22, nc;\n\t"

      "addc.cc.u64 t, %7, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %7, %14, %23, t;\n\t"
      "madc.hi.u64 %8, %14, %23, nc;\n\t"



      "mad.lo.cc.u64 %3, %15, %18, %3;\n\t"
      "madc.hi.cc.u64 c, %15, %18, 0;\n\t"
      
      "addc.cc.u64 t, %4, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %4, %15, %19, t;\n\t"
      "madc.hi.cc.u64 c, %15, %19, nc;\n\t"
      
      "addc.cc.u64 t, %5, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %5, %15, %20, t;\n\t"
      "madc.hi.cc.u64 c, %15, %20, nc;\n\t"
      
      "addc.cc.u64 t, %6, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %6, %15, %21, t;\n\t"
      "madc.hi.cc.u64 c, %15, %21, nc;\n\t"
      
      "addc.cc.u64 t, %7, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %7, %15, %22, t;\n\t"
      "madc.hi.cc.u64 c, %15, %22, nc;\n\t"
      
      "addc.cc.u64 t, %8, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %8, %15, %23, t;\n\t"
      "madc.hi.u64 %9, %15, %23, nc;\n\t"
      



      "mad.lo.cc.u64 %4, %16, %18, %4;\n\t"
      "madc.hi.cc.u64 c, %16, %18, 0;\n\t"
      
      "addc.cc.u64 t, %5, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %5, %16, %19, t;\n\t"
      "madc.hi.cc.u64 c, %16, %19, nc;\n\t"
      
      "addc.cc.u64 t, %6, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %6, %16, %20, t;\n\t"
      "madc.hi.cc.u64 c, %16, %20, nc;\n\t"
      
      "addc.cc.u64 t, %7, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %7, %16, %21, t;\n\t"
      "madc.hi.cc.u64 c, %16, %21, nc;\n\t"
      
      "addc.cc.u64 t, %8, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %8, %16, %22, t;\n\t"
      "madc.hi.cc.u64 c, %16, %22, nc;\n\t"
      
      "addc.cc.u64 t, %9, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %9, %16, %23, t;\n\t"
      "madc.hi.u64 %10, %16, %23, nc;\n\t"
      


      "mad.lo.cc.u64 %5, %17, %18, %5;\n\t"
      "madc.hi.cc.u64 c, %17, %18, 0;\n\t"
      
      "addc.cc.u64 t, %6, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %6, %17, %19, t;\n\t"
      "madc.hi.cc.u64 c, %17, %19, nc;\n\t"
      
      "addc.cc.u64 t, %7, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %7, %17, %20, t;\n\t"
      "madc.hi.cc.u64 c, %17, %20, nc;\n\t"
      
      "addc.cc.u64 t, %8, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %8, %17, %21, t;\n\t"
      "madc.hi.cc.u64 c, %17, %21, nc;\n\t"
      
      "addc.cc.u64 t, %9, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %9, %17, %22, t;\n\t"
      "madc.hi.cc.u64 c, %17, %22, nc;\n\t"
      
      "addc.cc.u64 t, %10, c;\n\t"
      "addc.u64 nc, 0, 0;\n\t"
      "mad.lo.cc.u64 %10, %17, %23, t;\n\t"
      "madc.hi.u64 %11, %17, %23, nc;\n\t"

      "}"
      : "+l"(r[0]),
      "+l"(r[1]),
      "+l"(r[2]),
      "+l"(r[3]),
      "+l"(r[4]),
      "+l"(r[5]),
      "+l"(r[6]),
      "+l"(r[7]),
      "+l"(r[8]),
      "+l"(r[9]),
      "+l"(r[10]),
      "+l"(r[11])
      : "l"(a[0]),
      "l"(a[1]),
      "l"(a[2]),
      "l"(a[3]),
      "l"(a[4]),
      "l"(a[5]),
      "l"(b[0]),
      "l"(b[1]),
      "l"(b[2]),
      "l"(b[3]),
      "l"(b[4]),
      "l"(b[5])
    );

    #pragma unroll
    for(int i = 0; i < 12; i++){
        c[i] = r[i];
    }
}


__global__ void kernel_msm_mont_mul(
    uint64_t *a, uint64_t *b, uint64_t *c, uint64_t *p, uint64_t inv){
    mul_mont_384((limb_t*)c, (limb_t*)a, (limb_t*)b, (limb_t*)p, (limb_t)inv); 
}

__global__ void kernel_cgbn_mul(
    cgbn_error_report_t* report, 
    uint32_t *a, uint32_t *b, uint32_t *c){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int instance = tid / TPI;
  context_t bn_context(cgbn_report_monitor, report, instance);
  env_t          bn_env(bn_context.env<env_t>());  

  env_t::cgbn_t dev_a, dev_b;
  env_t::cgbn_wide_t dev_c;
  cgbn_load(bn_env, dev_a, a);
  cgbn_load(bn_env, dev_b, b);

  cgbn_mul_wide(bn_env, dev_c, dev_a, dev_b);
  cgbn_store(bn_env, c, dev_c._low);
  cgbn_store(bn_env, c + 12, dev_c._high);
}

__global__ void kernel_cgbn_mont_mul(
    cgbn_error_report_t* report, 
    uint32_t *a, uint32_t *b, uint32_t *c, uint32_t *p, uint64_t inv){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int instance = tid / TPI;
  context_t bn_context(cgbn_report_monitor, report, instance);
  env_t          bn_env(bn_context.env<env_t>());  


  __shared__ uint32_t res[NUM * 3], buffer[NUM];
  
  DevFp dev_a, dev_b, dev_c, dev_p;
  cgbn_load(bn_env, dev_a.mont, a);
  cgbn_load(bn_env, dev_b.mont, b);
  cgbn_load(bn_env, dev_p.mont, p);
  dev_c = dev_a.mul(bn_env, dev_b, res, buffer, dev_p.mont, inv);
  cgbn_store(bn_env, c, dev_c.mont);

  //uint32_t np0=cgbn_bn2mont(bn_env, dev_a.mont, dev_a.mont, dev_p.mont);
  //cgbn_bn2mont(bn_env, dev_b.mont, dev_b.mont, dev_p.mont);
  //cgbn_mont_mul(bn_env, dev_c.mont, dev_a.mont, dev_b.mont, dev_p.mont, np0);
  //cgbn_mont2bn(bn_env, dev_c.mont, dev_c.mont, dev_p.mont, np0);

  
  //env_t::cgbn_wide_t tc;
  //cgbn_mul_wide(bn_env, tc, dev_a.mont, dev_b.mont);
  //__shared__ uint32_t cache_c[24];
  //__shared__ uint32_t cache_cc[12];
  //cgbn_store(bn_env, cache_c, tc._low);
  //cgbn_store(bn_env, cache_c + 12, tc._high);
  //if(threadIdx.x == 0){
  //    mont_384((limb_t*)cache_cc, (limb_t*)cache_c, (limb_t*)p, (limb_t)inv, false); 
  //    //uint64_t* c64 = (uint64_t *)c;
  //    //c64[0] = ((uint64_t*)cache_cc)[0];
  //    //c64[1] = ((uint64_t*)cache_cc)[1];
  //    //c64[2] = ((uint64_t*)cache_cc)[2];
  //    //c64[3] = ((uint64_t*)cache_cc)[3];
  //    //c64[4] = ((uint64_t*)cache_cc)[4];
  //    //c64[5] = ((uint64_t*)cache_cc)[5];
  //}
  /////__syncthreads();
  ////c[threadIdx.x] = cache_cc[threadIdx.x];
  //cgbn_load(bn_env, dev_c.mont, cache_cc);
  //if(cgbn_compare(bn_env, dev_c.mont, dev_p.mont) >= 0){
  //  cgbn_sub(bn_env, tc._low, dev_c.mont, dev_p.mont);
  //  cgbn_store(bn_env, c, tc._low);
  //}else{
  //  cgbn_store(bn_env, c, dev_c.mont);
  //}

}

__global__ void kernel_cgbn_mont_mul2(
    uint32_t *a, uint32_t *b, uint32_t *c, uint32_t *p, uint64_t inv){
   dev_mont_mul((uint64_t*)a, (uint64_t*)b, (uint64_t*)p, inv, (uint64_t*)c);  
}

void mpz_mul(mp_limb_t a[4], mp_limb_t b[4], mp_limb_t p[4], mp_limb_t inv, mp_limb_t c[4]){
printf("\n");
    const int n = 4;
	mp_limb_t res[2*n];
	mpn_mul_n(res, a, b, n);
	/*
	   The Montgomery reduction here is based on Algorithm 14.32 in
	   Handbook of Applied Cryptography
	   <http://cacr.uwaterloo.ca/hac/about/chap14.pdf>.
	 */

	for (size_t i = 0; i < n; ++i)
	{
		mp_limb_t k = inv * res[i];
        printf("k = %lu\n", k);
		/* calculate res = res + k * mod * b^i */
		//mp_limb_t carryout = mpn_addmul_1(res+i, p, n, k);
        mp_limb_t tmp[n];
        mp_limb_t carry1 = mpn_mul_1(tmp, p, n, k);
        printf("mul_1 %lu %u %u:", carry1, (uint32_t)carry1, (carry1>>32));
        for(int j = 0; j < n; j++){
            printf("%lu ", tmp[j]);
        }
        printf("\n");
        mp_limb_t carry2 = mpn_add_n(res+i, res+i, tmp, n);
        mp_limb_t carryout = carry1+carry2;
        printf("add %lu:", carry2);
        for(int j = 0; j < n; j++){
            printf("%lu ", res[i+j]);
        }
        printf("\n");
        printf("carry=%lu\n", carryout);
		carryout = mpn_add_1(res+n+i, res+n+i, n-i, carryout);
        for(int j = 0; j < n-i; j++){
            printf("%lu ", res[n+i+j]);
        }
        printf("\n");
	}

	if (mpn_cmp(res+n, p, n) >= 0)
	{
		const mp_limb_t borrow = mpn_sub(res+n, res+n, n, p, n);
	}

	mpn_copyi(c, res+n, n);
}

int main(){
    const int N = 4;
    uint64_t a[N], b[N], c1[N*2], c2[N*2];
    uint64_t p[4] = {
      TO_LIMB_T(0x8508c00000000001), TO_LIMB_T(0x170b5d4430000000),
        //TO_LIMB_T(0x1ef3622fba094800), TO_LIMB_T(0x1a22d9f300f5138f),
          TO_LIMB_T(0xc63b05c06ca1493b), TO_LIMB_T(0x1ae3a4617c510ea)
    };
    for(int i = 0; i < N; i++){
        a[i] = i;
        b[i] = i*10 + i;
    }
    uint64_t *dev_a, *dev_b, *dev_c1, *dev_c2, *dev_p;
    uint64_t inv = 0x8508bfffffffffff;

    hipMalloc((void**)&dev_a, sizeof(int64_t) * N);
    hipMalloc((void**)&dev_b, sizeof(int64_t) * N);
    hipMalloc((void**)&dev_c1, sizeof(int64_t) * N*2);
    hipMalloc((void**)&dev_c2, sizeof(int64_t) * N*2);
    hipMalloc((void**)&dev_p, sizeof(int64_t) * N);

    hipMemcpy(dev_a, a, sizeof(int64_t) * N, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(int64_t) * N, hipMemcpyHostToDevice);
    hipMemcpy(dev_p, p, sizeof(int64_t) * N, hipMemcpyHostToDevice);

    const int iters = 100;
    cgbn_error_report_t* report = nullptr;
    cgbn_error_report_alloc(&report); 

    for(int i = 0; i < iters; i++){
        kernel_msm_mul<<<1, 1>>>(dev_a, dev_b, dev_c1);
        kernel_cgbn_mul<<<1, TPI>>>(report, (uint32_t*)dev_a, (uint32_t*)dev_b, (uint32_t*)dev_c2);
    }
    hipMemcpy(c1, dev_c1, 2*N * sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemcpy(c2, dev_c2, 2*N * sizeof(int64_t), hipMemcpyDeviceToHost);

    //int cmp = memcmp(c1, c2, 2*N*sizeof(int64_t));
    //if(cmp != 0){
    //    for(int i = 0; i < 2*N; i++){
    //        printf("(%lu,%lu) ", c1[i], c2[i]);
    //    }
    //    printf("\n");
    //}else{
    //    printf("compare success\n");
    //}

    //for(int i = 0; i < iters; i++)
    //    kernel_msm_mont_mul<<<1, 1>>>(dev_a, dev_b, dev_c1, dev_p, inv);

    for(int i = 0; i < iters; i++)
        //kernel_cgbn_mont_mul<<<1, TPI>>>(report, (uint32_t*)dev_a, (uint32_t*)dev_b, (uint32_t*)dev_c2, (uint32_t*)dev_p, inv);
        kernel_cgbn_mont_mul2<<<1, 1>>>((uint32_t*)dev_a, (uint32_t*)dev_b, (uint32_t*)dev_c2, (uint32_t*)dev_p, inv);

    hipMemcpy(c1, dev_c1, N * sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemcpy(c2, dev_c2, N * sizeof(int64_t), hipMemcpyDeviceToHost);

    uint64_t c3[4];
    mpz_mul(a, b, p, inv, c3);
    int cmp = memcmp(c3, c2, N*sizeof(int64_t));
    if(cmp != 0){
        for(int i = 0; i < N; i++){
            printf("(%lu,%lu, %lu) ", c1[i], c2[i], c3[i]);
        }
        printf("\n");
    }else{
        printf("compare success\n");
    }
    return 0;
}
