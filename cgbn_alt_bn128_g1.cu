#include "hip/hip_runtime.h"

#include "cgbn_alt_bn128_g1.h"
#include "cgbn_alt_bn128_g1.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#include "cgbn/cgbn.h"
#include "utility/cpu_support.h"
#include "utility/cpu_simple_bn_math.h"
#include "utility/gpu_support.h"

namespace gpu{

alt_bn128_g1::alt_bn128_g1(const int count){
  init(count);
}
void alt_bn128_g1::init(const int count){
  x.init(count);
  y.init(count);
  z.init(count);
}
void alt_bn128_g1::init_host(const int count){
  x.init_host(count);
  y.init_host(count);
  z.init_host(count);
}
void alt_bn128_g1::release(){
  x.release();
  y.release();
  z.release();
}
void alt_bn128_g1::release_host(){
  x.release_host();
  y.release_host();
  z.release_host();
}
void alt_bn128_g1::copy_from_cpu(const alt_bn128_g1& g1){
  x.copy_from_cpu(g1.x);
  y.copy_from_cpu(g1.y);
  z.copy_from_cpu(g1.z);
}
void alt_bn128_g1::copy_to_cpu(alt_bn128_g1& g1){
  g1.x.copy_to_cpu(x);
  g1.y.copy_to_cpu(y);
  g1.z.copy_to_cpu(z);
}
void alt_bn128_g1::clear(CudaStream stream ){
  this->x.clear(stream);
  this->y.clear(stream);
  this->z.clear(stream);
}

__global__ void kernel_alt_bn128_g1_add(cgbn_error_report_t* report, alt_bn128_g1 a, alt_bn128_g1 b, alt_bn128_g1 c, const uint32_t count, cgbn_mem_t<BITS>* max_value, cgbn_mem_t<BITS>* modulus, const uint64_t inv){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int instance = tid / TPI;
  int local_instance = threadIdx.x / TPI;
  if(instance >= count) return;

  context_t bn_context(cgbn_report_monitor, report, instance);
  env_t          bn_env(bn_context.env<env_t>());  

  DevAltBn128G1 dev_a, dev_b;
  dev_a.load(bn_env, a, instance);
  dev_b.load(bn_env, b, instance);

  //const int n = BITS / 32;
  __shared__ uint32_t cache[64 * 8 * 3];
  uint32_t *res = &cache[local_instance * 8 * 3];
  //uint32_t *res = tmp_res + instance * 3 * n;
  //cgbn_mem_t<BITS>* buffer = tmp_buffer + instance;
  __shared__ uint32_t cache_buffer[64 * 8];
  uint32_t *buffer = &cache_buffer[local_instance * 8];
  env_t::cgbn_t local_max_value, local_modulus;
  cgbn_load(bn_env, local_max_value, max_value);
  cgbn_load(bn_env, local_modulus, modulus);

  DevAltBn128G1 dev_c;
  dev_alt_bn128_g1_add(bn_env, dev_a, dev_b, &dev_c, res, buffer, local_max_value, local_modulus, inv);
  dev_c.store(bn_env, c, instance);
}

__global__ void kernel_alt_bn128_g1_reduce_sum(
    cgbn_error_report_t* report, 
    alt_bn128_g1 values, 
    Fp_model scalars,
    const size_t *index_it,
    alt_bn128_g1 partial, 
    uint32_t* counters, 
    const int ranges_size, 
    const uint32_t *firsts,
    const uint32_t *seconds,
    cgbn_mem_t<BITS>* max_value,
    alt_bn128_g1 t_zero,
    alt_bn128_g1 t_one,
    Fp_model field_zero,
    Fp_model field_one,
    char *density,
    cgbn_mem_t<BITS>* bn_exponents,
    cgbn_mem_t<BITS>* modulus, const uint64_t inv
    ){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int instance = tid / TPI;
  if(instance >= ranges_size) return;

  context_t bn_context(cgbn_report_monitor, report, instance);
  env_t          bn_env(bn_context.env<env_t>());  

  const int n = BITS / 32;
  //uint32_t *res = tmp_res + instance * 3 * n;
  __shared__ uint32_t cache[64 * 3 * BITS/32];
  uint32_t *res = &cache[instance * 3 * n];
  //cgbn_mem_t<BITS>* buffer = tmp_buffer + instance;
  __shared__ uint32_t cache_buffer[64 * BITS/32];
  uint32_t *buffer = &cache_buffer[instance * BITS/32];
  env_t::cgbn_t local_max_value, local_modulus;
  cgbn_load(bn_env, local_max_value, max_value);
  cgbn_load(bn_env, local_modulus, modulus);

  DevAltBn128G1 result, dev_t_zero;
  DevFp dev_field_zero, dev_field_one;
  dev_t_zero.load(bn_env, t_zero, 0);
  dev_field_zero.load(bn_env, field_zero, 0);
  dev_field_one.load(bn_env, field_one, 0);
  result.copy_from(bn_env, dev_t_zero);
  int count = 0;
  for(int i = firsts[instance]; i < seconds[instance]; i++){
    const int j = index_it[i];
    DevFp scalar;
    scalar.load(bn_env, scalars, j);
    if(scalar.isequal(bn_env, dev_field_zero)){
    }
    else if(scalar.isequal(bn_env, dev_field_one)){
      DevAltBn128G1 dev_b;
      dev_b.load(bn_env, values, i);
      dev_alt_bn128_g1_add(bn_env, result, dev_b, &result, res, buffer, local_max_value, local_modulus, inv);
    }
    else{
      const int group_thread = threadIdx.x & (TPI-1);
      if(group_thread == 0){
        density[i] = 1;
      }
      //DevFp a = scalar.as_bigint(bn_env, res, buffer, local_modulus, inv);
      //a.store(bn_env, bn_exponents, i);
      count += 1;
    }
  }  result.store(bn_env, partial, instance);
  const int group_thread = threadIdx.x & (TPI-1);
  if(group_thread == 0)
    counters[instance] = count;
}

__global__ void kernel_alt_bn128_g1_reduce_sum_one_range_pre(
    cgbn_error_report_t* report, 
    Fp_model scalars,
    const size_t *index_it,
    uint32_t* counters, 
    char* flags,
    const int ranges_size, 
    const uint32_t* firsts,
    const uint32_t* seconds,
    cgbn_mem_t<BITS>* max_value,
    Fp_model field_zero,
    Fp_model field_one,
    char *density,
    cgbn_mem_t<BITS>* bn_exponents,
    const uint64_t inv,
    cgbn_mem_t<BITS>* field_modulus, const uint64_t field_inv
    ){
  int local_instance = threadIdx.x / TPI;//0~63
  int local_instances = 64;
  int instance = blockIdx.x * local_instances + local_instance;

  int range_offset = blockIdx.y * gridDim.x * local_instances;
  int first = firsts[blockIdx.y];
  int second = seconds[blockIdx.y];
  int reduce_depth = second - first;//30130

  context_t bn_context(cgbn_report_monitor, report, range_offset + instance);
  env_t          bn_env(bn_context.env<env_t>());  

  __shared__ uint32_t cache_res[64 * 24];
  uint32_t *res = &cache_res[local_instance * 24];
  __shared__ uint32_t cache_buffer[512];
  uint32_t *buffer = &cache_buffer[local_instance * 8];
  env_t::cgbn_t local_field_modulus;
  cgbn_load(bn_env, local_field_modulus, field_modulus);

  DevFp dev_field_zero, dev_field_one;
  dev_field_zero.load(bn_env, field_zero, 0);
  dev_field_one.load(bn_env, field_one, 0);
  int count = 0;
  for(int i = first + instance; i < first + reduce_depth; i+= gridDim.x * local_instances){
    const int j = index_it[i];
    DevFp scalar;
    scalar.load(bn_env, scalars, j);
    if(scalar.isequal(bn_env, dev_field_zero)){
    }
    else if(scalar.isequal(bn_env, dev_field_one)){
      flags[j] = 1;
    }
    else{
      const int group_thread = threadIdx.x & (TPI-1);
      if(group_thread == 0){
        density[i] = 1;
      }
      DevFp a = scalar.as_bigint(bn_env, res, buffer, local_field_modulus, field_inv);
      a.store(bn_env, bn_exponents, i);
      count += 1;
    }
  }
  __shared__ int cache_counters[64];
  const int group_thread = threadIdx.x & (TPI-1);
  if(group_thread == 0)
    cache_counters[local_instance] = count;
  __syncthreads();
  if(local_instance == 0){
    for(int i = 1; i < local_instances; i++){
      count += cache_counters[i];
    }
    if(group_thread == 0){
      counters[blockIdx.y * gridDim.x + blockIdx.x] = count;
    }
  }
}
__global__ void kernel_alt_bn128_g1_reduce_sum_one_range(
    cgbn_error_report_t* report, 
    alt_bn128_g1 values, 
    Fp_model scalars,
    const size_t *index_it,
    alt_bn128_g1 partial, 
    const int ranges_size, 
    const uint32_t* firsts,
    const uint32_t* seconds,
    const char* flags,
    cgbn_mem_t<BITS>* max_value,
    alt_bn128_g1 t_zero,
    cgbn_mem_t<BITS>* modulus, const uint64_t inv
    ){
  int local_instance = threadIdx.x / TPI;//0~63
  int local_instances = 64;
  int instance = blockIdx.x * local_instances + local_instance;

  int range_offset = blockIdx.y * gridDim.x * local_instances;
  int first = firsts[blockIdx.y];
  int second = seconds[blockIdx.y];
  int reduce_depth = second - first;//30130

  context_t bn_context(cgbn_report_monitor, report, range_offset + instance);
  env_t          bn_env(bn_context.env<env_t>());  

  __shared__ uint32_t cache_res[64 * 24];
  uint32_t *res = &cache_res[local_instance * 24];
  __shared__ uint32_t cache_buffer[512];
  uint32_t *buffer = &cache_buffer[local_instance * 8];
  env_t::cgbn_t local_max_value, local_modulus;
  cgbn_load(bn_env, local_max_value, max_value);
  cgbn_load(bn_env, local_modulus, modulus);

  DevAltBn128G1 result;
  DevFp dev_field_zero, dev_field_one;
  result.load(bn_env, t_zero, 0);
  for(int i = first + instance; i < first + reduce_depth; i+= gridDim.x * local_instances){
    const int j = index_it[i];
    if(flags[j] == 1){
      DevAltBn128G1 dev_b;
      dev_b.load(bn_env, values, i);
      dev_alt_bn128_g1_add(bn_env, result, dev_b, &result, res, buffer, local_max_value, local_modulus, inv);
    }
  }
  result.store(bn_env, partial, range_offset + instance);
  __syncthreads();
  if(local_instance == 0){
    for(int i = 1; i < local_instances; i++){
      DevAltBn128G1 dev_b;
      dev_b.load(bn_env, partial, range_offset + instance + i);
      dev_alt_bn128_g1_add(bn_env, result, dev_b, &result, res, buffer, local_max_value, local_modulus, inv);
    }
    result.store(bn_env, partial, range_offset + instance);
  }
}

__global__ void kernel_alt_bn128_g1_reduce_sum_one_range2(
    cgbn_error_report_t* report, 
    alt_bn128_g1 values, 
    Fp_model scalars,
    const size_t *index_it,
    alt_bn128_g1 partial, 
    const int ranges_size, 
    const uint32_t* firsts,
    const uint32_t* seconds,
    const char* flags,
    cgbn_mem_t<BITS>* max_value,
    alt_bn128_g1 t_zero,
    cgbn_mem_t<BITS>* modulus, const uint64_t inv
    ){
  int local_instance = threadIdx.x / TPI;//0~63
  int local_instances = 64;
  int instance = blockIdx.x * local_instances + local_instance;

  int range_offset = blockIdx.y * gridDim.x * local_instances;
  int first = firsts[blockIdx.y];
  int second = seconds[blockIdx.y];
  int reduce_depth = second - first;//30130

  context_t bn_context(cgbn_report_monitor, report, range_offset + instance);
  env_t          bn_env(bn_context.env<env_t>());  

  __shared__ uint32_t cache_res[64 * 24];
  uint32_t *res = &cache_res[local_instance * 24];
  __shared__ uint32_t cache_buffer[512];
  uint32_t *buffer = &cache_buffer[local_instance * 8];
  env_t::cgbn_t local_max_value, local_modulus;
  cgbn_load(bn_env, local_max_value, max_value);
  cgbn_load(bn_env, local_modulus, modulus);

  DevAltBn128G1 result;
  result.load(bn_env, t_zero, 0);
  for(int i = first + instance; i < first + reduce_depth; i+= gridDim.x * local_instances){
    const int j = index_it[i];
    if(flags[j] == 1){
      DevAltBn128G1 dev_b;
      dev_b.load(bn_env, values, i);
      dev_alt_bn128_g1_add(bn_env, result, dev_b, &result, res, buffer, local_max_value, local_modulus, inv);
    }
  }
  result.store(bn_env, partial, range_offset + instance);
}

__global__ void kernel_alt_bn128_g1_reduce_sum_one_range3(
    cgbn_error_report_t* report, 
    alt_bn128_g1 partial, 
    const int n, 
    const int range_instances,
    cgbn_mem_t<BITS>* max_value,
    cgbn_mem_t<BITS>* modulus, const uint64_t inv
    ){
  int local_instance = threadIdx.x / TPI;//0~63
  int local_instances = blockDim.x / TPI;
  int instance = blockIdx.x * local_instances + local_instance;

  int range_offset = blockIdx.y * range_instances;

  context_t bn_context(cgbn_report_monitor, report, range_offset + instance);
  env_t          bn_env(bn_context.env<env_t>());  

  __shared__ uint32_t cache_res[64 * 24];
  uint32_t *res = &cache_res[local_instance * 24];
  __shared__ uint32_t cache_buffer[512];
  uint32_t *buffer = &cache_buffer[local_instance * 8];
  env_t::cgbn_t local_max_value, local_modulus;
  cgbn_load(bn_env, local_max_value, max_value);
  cgbn_load(bn_env, local_modulus, modulus);

  DevAltBn128G1 result;
  result.load(bn_env, partial, range_offset + instance);
  int i = instance + gridDim.x * local_instances;

  if(i < n){
    DevAltBn128G1 dev_b;
    dev_b.load(bn_env, partial, range_offset + i);
    dev_alt_bn128_g1_add(bn_env, result, dev_b, &result, res, buffer, local_max_value, local_modulus, inv);
  }
  result.store(bn_env, partial, range_offset + instance);
}

__global__ void kernel_alt_bn128_g1_reduce_sum_one_range4(
    cgbn_error_report_t* report, 
    alt_bn128_g1 partial, 
    const int n, 
    const int range_offset,
    cgbn_mem_t<BITS>* max_value,
    cgbn_mem_t<BITS>* modulus, const uint64_t inv
    ){
  int instance = threadIdx.x / TPI;

  context_t bn_context(cgbn_report_monitor, report, instance);
  env_t          bn_env(bn_context.env<env_t>());  

  __shared__ uint32_t res[24];
  __shared__ uint32_t buffer[8];
  env_t::cgbn_t local_max_value, local_modulus;
  cgbn_load(bn_env, local_max_value, max_value);
  cgbn_load(bn_env, local_modulus, modulus);

  DevAltBn128G1 result;
  result.load(bn_env, partial, 0);

  for(int i = 1; i < n; i++){
    DevAltBn128G1 dev_b;
    dev_b.load(bn_env, partial, i * range_offset);
    dev_alt_bn128_g1_add(bn_env, result, dev_b, &result, res, buffer, local_max_value, local_modulus, inv);
  }
  result.store(bn_env, partial, 0);
}

__global__ void kernel_alt_bn128_g1_reduce_sum(
    cgbn_error_report_t* report, 
    alt_bn128_g1 partial_in, 
    const uint32_t* counters_in, 
    alt_bn128_g1 partial_out, 
    uint32_t* counters_out, 
    const int ranges_size, 
    cgbn_mem_t<BITS>* max_value,
    alt_bn128_g1 t_zero,
    int depth, int step,
    cgbn_mem_t<BITS>* modulus, const uint64_t inv
    ){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int instance = tid / TPI;
  if(instance >= ranges_size) return;

  context_t bn_context(cgbn_report_monitor, report, instance);
  env_t          bn_env(bn_context.env<env_t>());  

  //const int n = BITS / 32;
  __shared__ uint32_t cache_res[64 * 3 * BITS/32];
  uint32_t *res = &cache_res[instance * BITS/32 * 3];
  //uint32_t *res = tmp_res + instance * 3 * n;
  //cgbn_mem_t<BITS>* buffer = tmp_buffer + instance;
  __shared__ uint32_t cache_buffer[64 * BITS/32];
  uint32_t *buffer = &cache_buffer[instance * BITS/32];
  env_t::cgbn_t local_max_value, local_modulus;
  cgbn_load(bn_env, local_max_value, max_value);
  cgbn_load(bn_env, local_modulus, modulus);

  DevAltBn128G1 result;
  result.load(bn_env, t_zero, 0);
  //result.copy_from(bn_env, dev_t_zero);
  int count = 0;
  for(int i = 0; i < depth; i++){
    DevAltBn128G1 dev_b;
    dev_b.load(bn_env, partial_in, instance * depth * step + i * step);
    dev_alt_bn128_g1_add(bn_env, result, dev_b, &result, res, buffer, local_max_value, local_modulus, inv);
    count += counters_in[instance * depth + i];
  }
  result.store(bn_env, partial_out, instance);
  const int group_thread = threadIdx.x & (TPI-1);
  if(group_thread == 0){
    counters_out[instance] = count;
  }
}

__global__ void kernel_alt_bn128_g1_reduce_sum2(
    cgbn_error_report_t* report, 
    alt_bn128_g1 data, 
    alt_bn128_g1 out, 
    const int n, 
    cgbn_mem_t<BITS>* max_value,
    cgbn_mem_t<BITS>* modulus, const uint64_t inv
    ){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int instance = tid / TPI;
  int local_instance = threadIdx.x / TPI;
  int local_instances = blockDim.x / TPI;
  int offset = gridDim.x * local_instances;
  if(instance >= n) return;

  context_t bn_context(cgbn_report_monitor, report, instance);
  env_t          bn_env(bn_context.env<env_t>());  

  __shared__ uint32_t cache_res[64 * 24];
  uint32_t *res = &cache_res[local_instance * 24];
  __shared__ uint32_t cache_buffer[64 * 8];
  uint32_t *buffer = &cache_buffer[local_instance * 8];
  env_t::cgbn_t local_max_value, local_modulus;
  cgbn_load(bn_env, local_max_value, max_value);
  cgbn_load(bn_env, local_modulus, modulus);

  DevAltBn128G1 result;
  result.load(bn_env, data, instance);
  for(int i = instance + offset; i < n; i+=offset){
    DevAltBn128G1 dev_b;
    dev_b.load(bn_env, data, i);
    dev_alt_bn128_g1_add(bn_env, result, dev_b, &result, res, buffer, local_max_value, local_modulus, inv);
  }
  result.store(bn_env, out, instance);
  __syncthreads();
  if(local_instance == 0){
    for(int i = 1; i < local_instances; i++){
      DevAltBn128G1 dev_b;
      dev_b.load(bn_env, out, instance + i);
      dev_alt_bn128_g1_add(bn_env, result, dev_b, &result, res, buffer, local_max_value, local_modulus, inv);
    }
    result.store(bn_env, out, instance);
  }
}

int alt_bn128_g1_add(alt_bn128_g1 a, alt_bn128_g1 b, alt_bn128_g1 c, const uint32_t count, cgbn_mem_t<BITS>* max_value, cgbn_mem_t<BITS>* modulus, const uint64_t inv){
  cgbn_error_report_t *report;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 

  uint32_t instances = std::min(count, (uint32_t)max_threads_per_block);
  uint32_t threads = instances * TPI;
  uint32_t blocks = (count + instances - 1) / instances;

  kernel_alt_bn128_g1_add<<<blocks, threads>>>(report, a, b, c, count, max_value, modulus, inv);

  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);
  CUDA_CHECK(cgbn_error_report_free(report));
  return 0;
}

int alt_bn128_g1_reduce_sum(
    alt_bn128_g1 values, 
    Fp_model scalars, 
    const size_t *index_it,
    alt_bn128_g1 partial, 
    uint32_t *counters,
    const uint32_t ranges_size,
    const uint32_t *firsts,
    const uint32_t *seconds,
    cgbn_mem_t<BITS>* max_value,
    alt_bn128_g1 t_zero,
    alt_bn128_g1 t_one,
    Fp_model field_zero,
    Fp_model field_one,
    char *density,
    cgbn_mem_t<BITS>* bn_exponents,
    cgbn_mem_t<BITS>* modulus, const uint64_t inv
    ){
  cgbn_error_report_t *report;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 

  uint32_t instances = std::min(ranges_size, (uint32_t)max_threads_per_block);
  uint32_t threads = instances * TPI;
  uint32_t blocks = (ranges_size + instances - 1) / instances;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  kernel_alt_bn128_g1_reduce_sum<<<blocks, threads>>>(report, values, scalars, index_it, partial, counters, ranges_size, firsts, seconds, max_value, t_zero, t_one, field_zero, field_one, density, bn_exponents, modulus, inv);

  CUDA_CHECK(hipDeviceSynchronize());
  hipEventRecord(stop, 0);
  hipEventSynchronize(start); 
  hipEventSynchronize(stop);   
  float costtime;
  hipEventElapsedTime(&costtime, start, stop);
  printf("kernel time = %fms\n", costtime);
  CGBN_CHECK(report);
  CUDA_CHECK(cgbn_error_report_free(report));
  return 0;
}

int alt_bn128_g1_reduce_sum_one_range(
    alt_bn128_g1 values, 
    Fp_model scalars, 
    const size_t *index_it,
    alt_bn128_g1 partial, 
    uint32_t *counters,
    char* flags,
    const uint32_t ranges_size,
    const uint32_t *firsts,
    const uint32_t *seconds,
    cgbn_mem_t<BITS>* max_value,
    alt_bn128_g1 t_zero,
    Fp_model field_zero,
    Fp_model field_one,
    char *density,
    cgbn_mem_t<BITS>* bn_exponents,
    cgbn_mem_t<BITS>* modulus, const uint64_t inv,
    cgbn_mem_t<BITS>* field_modulus, const uint64_t field_inv,
    const int max_reduce_depth
    ){
  cgbn_error_report_t *report = get_error_report();

  uint32_t threads = 512;
  //const int reduce_depth = 30130;//second - first;
  const int local_instances = 64 * BlockDepth;
  uint32_t block_x =  (max_reduce_depth + local_instances - 1) / local_instances;
  dim3 blocks(block_x, ranges_size, 1);
  kernel_alt_bn128_g1_reduce_sum_one_range_pre<<<blocks, threads>>>(report, scalars, index_it, counters, flags, ranges_size, firsts, seconds, max_value, field_zero, field_one, density, bn_exponents, inv, field_modulus, field_inv);

//*********test
  const int local_instances2 = 64 * BlockDepth;
  uint32_t block_x2 =  (max_reduce_depth + local_instances2 - 1) / local_instances2;
  dim3 blocks2(block_x2, ranges_size, 1);
  kernel_alt_bn128_g1_reduce_sum_one_range2<<<blocks2, threads>>>(report, values, scalars, index_it, partial, ranges_size, firsts, seconds, flags, max_value, t_zero, modulus, inv);
  int n = block_x2 * 64;
  int range_offset = n;
  while(n >= 2){
    int instances = std::min(64, n/2);
    int threads = instances * TPI; 
    int blockx = n / (instances * 2);
    kernel_alt_bn128_g1_reduce_sum_one_range3<<<dim3(blockx, ranges_size, 1), threads>>>(report, partial, n, range_offset, max_value, modulus, inv);
    n /= 2;
  }
  kernel_alt_bn128_g1_reduce_sum_one_range4<<<1, TPI>>>(report, partial, ranges_size, range_offset, max_value, modulus, inv);
//********test

  //kernel_alt_bn128_g1_reduce_sum_one_range<<<blocks, threads>>>(report, values, scalars, index_it, partial, ranges_size, firsts, seconds, flags, max_value, t_zero, modulus, inv);
  //CUDA_CHECK(hipDeviceSynchronize());
  //CGBN_CHECK(report);
  return 0;
}
void alt_bn128_g1_reduce_sum(
    alt_bn128_g1 partial_in, 
    const uint32_t *counters_in,
    alt_bn128_g1 partial_out, 
    uint32_t *counters_out,
    const uint32_t ranges_size,
    cgbn_mem_t<BITS>* max_value,
    alt_bn128_g1 t_zero,
    cgbn_mem_t<BITS>* modulus, const uint64_t inv,
    const int max_reduce_depth
    ){
  cgbn_error_report_t *report = get_error_report();
  uint32_t instances = std::min(ranges_size, (uint32_t)max_threads_per_block);
  uint32_t threads = instances * TPI;
  uint32_t blocks = (ranges_size + instances - 1) / instances;
  //int reduce_depth = 30130;
  const int local_instances = 64 * BlockDepth;
  uint32_t depth =  (max_reduce_depth + local_instances - 1) / local_instances;
  int step = 64;
  kernel_alt_bn128_g1_reduce_sum<<<blocks, threads>>>(report, partial_in, counters_in, partial_out, counters_out, ranges_size, max_value, t_zero, depth, step, modulus, inv);
  //CUDA_CHECK(hipDeviceSynchronize());
  //CGBN_CHECK(report);
}

void alt_bn128_g1_reduce_sum_one_instance(
    alt_bn128_g1 partial_in, 
    const uint32_t *counters_in,
    alt_bn128_g1 partial_out, 
    uint32_t *counters_out,
    cgbn_mem_t<BITS>* max_value,
    alt_bn128_g1 t_zero,
    cgbn_mem_t<BITS>* modulus, const uint64_t inv,
    const int max_reduce_depth
    ){
  cgbn_error_report_t *report = get_error_report();
  uint32_t instances = 1;
  uint32_t threads = instances * TPI;
  uint32_t blocks = 1;
  kernel_alt_bn128_g1_reduce_sum<<<blocks, threads>>>(report, partial_in, counters_in, partial_out, counters_out, 1, max_value, t_zero, max_reduce_depth, 1, modulus, inv);
  CUDA_CHECK(hipDeviceSynchronize());
  //CGBN_CHECK(report);
}

template<int BlockSize, int BlockNum>
__global__ void test(
    cgbn_error_report_t* report, 
    alt_bn128_g1 data, 
    alt_bn128_g1 out, 
    int n,
    cgbn_mem_t<BITS>* max_value,
    cgbn_mem_t<BITS>* modulus, const uint64_t inv
    ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int instance = tid / TPI;
  int local_instance = threadIdx.x / TPI;
  context_t bn_context(cgbn_report_monitor, report, instance);
  env_t          bn_env(bn_context.env<env_t>());  
  DevAltBn128G1 a;
  a.load(bn_env, data, instance);
  __shared__ uint32_t cache_buffer[BlockSize*8];
  __shared__ uint32_t cache_res[BlockSize*24];
  uint32_t *buffer = &cache_buffer[local_instance * 8];
  uint32_t *res = &cache_res[local_instance * 24];
  env_t::cgbn_t local_max_value, local_modulus;
  cgbn_load(bn_env, local_max_value, max_value);
  cgbn_load(bn_env, local_modulus, modulus);
  for(int i = instance + BlockNum*BlockSize; i < n; i+=BlockNum*BlockSize){
    DevAltBn128G1 b;
    b.load(bn_env, data, i);
    dev_alt_bn128_g1_add(bn_env, a, b, &a, res, buffer, local_max_value, local_modulus, inv);
  }
  a.store(bn_env, out, instance);
  //__syncthreads();
  //if(local_instance == 0){
  //  for(int i = 1; i < BlockSize && i < n; i++){
  //    DevAltBn128G1 b;
  //    b.load(bn_env, out, instance + i);
  //    dev_alt_bn128_g1_add(bn_env, a, b, &a, res, buffer, local_max_value, local_modulus, inv);
  //  }
  //  a.store(bn_env, out, blockIdx.x);
  //}
}
void alt_bn128_g1_reduce_sum2(
    alt_bn128_g1 data, 
    alt_bn128_g1 out, 
    const uint32_t n,
    cgbn_mem_t<BITS>* max_value,
    cgbn_mem_t<BITS>* modulus, const uint64_t inv, 
    CudaStream stream){
  cgbn_error_report_t *report = get_error_report();
  uint32_t threads = 512;
  uint32_t local_instances = threads / TPI;//64
  uint32_t instances = std::min(n, (uint32_t)(local_instances * BlockDepth));
  //uint32_t blocks = (n + instances - 1) / instances;
  //kernel_alt_bn128_g1_reduce_sum2<<<blocks, threads>>>(report, data, out, n, max_value, modulus, inv);
  test<64, 64><<<64, 512, 0, stream>>>(report, data, out, n-1, max_value, modulus, inv);
  const int tmp_n = 64*64; 
  test<64, 8><<<8, 512, 0, stream>>>(report, out, data, tmp_n, max_value, modulus, inv);
  test<16, 4><<<4, 128, 0, stream>>>(report, data, out, 64*8, max_value, modulus, inv);
  test<8, 1><<<1, 64, 0, stream>>>(report, out, data, 64, max_value, modulus, inv);
  test<1, 1><<<1, 8, 0, stream>>>(report, data, out, 8, max_value, modulus, inv);
  //CUDA_CHECK(hipDeviceSynchronize());
}


void init_error_report(){
  get_error_report();
}

__global__ void kernel_warmup(){
  int sum = 0;
  for(int i = 0; i < 1000; i++){
    sum += i;
  }
  printf("warm up : %d\n", sum);
}
void warm_up(){
  kernel_warmup<<<1, 1>>>();
  CUDA_CHECK(hipDeviceSynchronize());
}

} //gpu
