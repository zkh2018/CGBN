#include "hip/hip_runtime.h"
#include "cgbn_alt_bn128_g1.h"
#include "cgbn_fp.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#include "cgbn/cgbn.h"
#include "utility/cpu_support.h"
#include "utility/cpu_simple_bn_math.h"
#include "utility/gpu_support.h"

namespace gpu{

alt_bn128_g1::alt_bn128_g1(const int count){
  init(count);
}
void alt_bn128_g1::init(const int count){
  x.init(count);
  y.init(count);
  z.init(count);
}
void alt_bn128_g1::init_host(const int count){
  x.init_host(count);
  y.init_host(count);
  z.init_host(count);
}
void alt_bn128_g1::release(){
  x.release();
  y.release();
  z.release();
}
void alt_bn128_g1::release_host(){
  x.release_host();
  y.release_host();
  z.release_host();
}
void alt_bn128_g1::copy_from_cpu(const alt_bn128_g1& g1){
  x.copy_from_cpu(g1.x);
  y.copy_from_cpu(g1.y);
  z.copy_from_cpu(g1.z);
}
void alt_bn128_g1::copy_to_cpu(alt_bn128_g1& g1){
  g1.x.copy_from_cpu(x);
  g1.y.copy_from_cpu(y);
  g1.z.copy_from_cpu(z);
}

struct DevFp{
  env_t::cgbn_t mont, modulus;
  uint64_t inv;
  inline __device__ bool is_zero(env_t& bn_env){
    return cgbn_equals_ui32(bn_env, mont, 0);
  }
  inline __device__ bool isequal(env_t& bn_env, const DevFp& other){
    return cgbn_equals(bn_env, mont, other.mont);
  }

  inline __device__ DevFp squared(env_t& bn_env, uint32_t *res, cgbn_mem_t<BITS>* tmp_buffer){
    device_mul_reduce(bn_env, res, mont, mont, modulus, tmp_buffer, inv);
    DevFp ret;
    ret.inv = inv;
    cgbn_set(bn_env, ret.modulus, modulus);
    cgbn_load(bn_env, ret.mont, res + BITS/32);
    return ret;
  }
  inline __device__ DevFp mul(env_t& bn_env, const DevFp& other, uint32_t *res, cgbn_mem_t<BITS>* tmp_buffer){
    device_mul_reduce(bn_env, res, mont, other.mont, modulus, tmp_buffer, inv);
    DevFp ret;
    ret.inv = inv;
    cgbn_set(bn_env, ret.modulus, modulus);
    cgbn_load(bn_env, ret.mont, res + BITS/32);
    return ret;

  }
  inline __device__ DevFp sub(env_t& bn_env, const DevFp& other, const env_t::cgbn_t& max_value){
    DevFp ret;
    device_fp_sub(bn_env, ret.mont, mont, other.mont, modulus, max_value);
    cgbn_set(bn_env, ret.modulus, modulus);
    ret.inv = inv;
    return ret;
  }
  inline __device__ DevFp add(env_t& bn_env, const DevFp& other, const env_t::cgbn_t& max_value){
    DevFp ret;
    device_fp_add(bn_env, ret.mont, mont, other.mont, modulus, max_value);
    cgbn_set(bn_env, ret.modulus, modulus);
    ret.inv = inv;
    return ret;
  }

  inline __device__ void print_array(env_t& bn_env, env_t::cgbn_t& data, cgbn_mem_t<BITS>* buffer){
    cgbn_store(bn_env, buffer, data);
    if(threadIdx.x == 0){
      for(int i = 0; i < BITS/32; i++){
        printf("%u ", buffer->_limbs[i]);
      }
      printf("\n");
    }
  }
  inline __device__ void print_array_64(env_t& bn_env, env_t::cgbn_t& data, cgbn_mem_t<BITS>* buffer){
    cgbn_store(bn_env, buffer, data);
    if(threadIdx.x == 0){
      uint64_t *p = (uint64_t*)buffer->_limbs;
      for(int i = 0; i < BITS/64; i++){
        printf("%lu ", p[i]);
      }
      printf("\n");
    }
  }
  inline __device__ void print(env_t& bn_env, cgbn_mem_t<BITS>* buffer){
    if(threadIdx.x== 0)
    printf("mont:\n");
    print_array(bn_env, mont, buffer);
    if(threadIdx.x== 0)
    printf("modulus:\n");
    print_array(bn_env, modulus, buffer);
    if(threadIdx.x== 0)
    printf("inv: %lu \n\n", inv, buffer);
  }
  inline __device__ void print_64(env_t& bn_env, cgbn_mem_t<BITS>* buffer){
    if(threadIdx.x== 0)
    printf("mont:\n");
    print_array_64(bn_env, mont, buffer);
    if(threadIdx.x== 0)
    printf("modulus:\n");
    print_array_64(bn_env, modulus, buffer);
    if(threadIdx.x== 0)
    printf("inv: %lu \n\n", inv, buffer);
  }
};


struct DevAltBn128G1{
  DevFp x, y, z;

  __device__ void load(env_t& bn_env, alt_bn128_g1& a, const int offset){
    cgbn_load(bn_env, x.mont, a.x.mont_repr_data + offset);
    cgbn_load(bn_env, x.modulus, a.x.modulus_data + offset);
    x.inv = a.x.inv;

    cgbn_load(bn_env, y.mont, a.y.mont_repr_data + offset);
    cgbn_load(bn_env, y.modulus, a.y.modulus_data + offset);
    y.inv = a.y.inv;

    cgbn_load(bn_env, z.mont, a.z.mont_repr_data + offset);
    cgbn_load(bn_env, z.modulus, a.z.modulus_data + offset);
    z.inv = a.z.inv;
  }
  __device__ void store(env_t& bn_env, alt_bn128_g1& a, const int offset){
    cgbn_store(bn_env, a.x.mont_repr_data + offset, x.mont);
    cgbn_store(bn_env, a.x.modulus_data + offset, x.modulus);
    a.x.inv = x.inv;

    cgbn_store(bn_env, a.y.mont_repr_data + offset, y.mont);
    cgbn_store(bn_env, a.y.modulus_data + offset, y.modulus);
    a.y.inv = y.inv;

    cgbn_store(bn_env, a.z.mont_repr_data + offset, z.mont);
    cgbn_store(bn_env, a.z.modulus_data + offset, z.modulus);
    a.z.inv = z.inv;
  }
  __device__ void store(env_t& bn_env, DevFp& x_, DevFp& y_, DevFp& z_, alt_bn128_g1& a, const int offset){
    cgbn_store(bn_env, a.x.mont_repr_data + offset, x_.mont);
    cgbn_store(bn_env, a.x.modulus_data + offset, x_.modulus);
    a.x.inv = x_.inv;

    cgbn_store(bn_env, a.y.mont_repr_data + offset, y_.mont);
    cgbn_store(bn_env, a.y.modulus_data + offset, y_.modulus);
    a.y.inv = y_.inv;

    cgbn_store(bn_env, a.z.mont_repr_data + offset, z_.mont);
    cgbn_store(bn_env, a.z.modulus_data + offset, z_.modulus);
    a.z.inv = z_.inv;
  }

  inline __device__ bool is_zero(env_t& bn_env){
    return z.is_zero(bn_env);
  }
  inline __device__ void dbl(env_t& bn_env, alt_bn128_g1& c, uint32_t* tmp_res, cgbn_mem_t<BITS>* tmp_buffer, env_t::cgbn_t& max_value, const int instance){
    if(is_zero(bn_env)){
      store(bn_env, c, instance);
      return;
    }

    const int n = BITS/32;
    uint32_t *res = tmp_res + instance * 3 * n;
    cgbn_mem_t<BITS>* buffer = tmp_buffer + instance;
    //A = squared(a.x)
    DevFp A = x.squared(bn_env, res, buffer);
    //B = squared(a.y)
    DevFp B = y.squared(bn_env, res, buffer);
    //C = squared(B)
    DevFp C = B.squared(bn_env, res, buffer);
    //D = squared(a.x + B) - A - C
    DevFp xb = x.add(bn_env, B, max_value);
    DevFp xb2 = xb.squared(bn_env, res, buffer);
    xb = xb2.sub(bn_env, A, max_value);
    DevFp tmp_D = xb.sub(bn_env, C, max_value);
    //D = D+D
    DevFp D = tmp_D.add(bn_env, tmp_D, max_value);
    //E = A + A + A
    DevFp A2 = A.add(bn_env, A, max_value);
    DevFp E = A2.add(bn_env, A, max_value);
    //F = squared(E)
    DevFp F = E.squared(bn_env, res, buffer);
    //X3 = F - (D+D)
    DevFp X3 = F.sub(bn_env, D.add(bn_env, D, max_value), max_value);
    //eightC = C+C
    DevFp eightC1 = C.add(bn_env, C, max_value);
    //eightC = eightC + eightC
    DevFp eightC2 = eightC1.add(bn_env, eightC1, max_value);
    //eightC = eightC + eightC
    DevFp eightC = eightC2.add(bn_env, eightC2, max_value);
    //Y3 = E * (D - X3) - eightC
    DevFp dx3 = D.sub(bn_env, X3, max_value);
    DevFp edx3 = E.mul(bn_env, dx3, res, buffer);
    DevFp Y3 = edx3.sub(bn_env, eightC, max_value);
    //Y1Z1 = (a.y * a.z)
    DevFp Y1Z1 = y.mul(bn_env, z, res, buffer);
    //Z3 = Y1Z1 + Y1Z1
    DevFp Z3 = Y1Z1.add(bn_env, Y1Z1, max_value);
    //c.x = X3, c.y = Y3, c.z = Z3
    store(bn_env, X3, Y3, Z3, c, instance);
  }
};

__global__ void kernel_alt_bn128_g1_add(cgbn_error_report_t* report, alt_bn128_g1 a, alt_bn128_g1 b, alt_bn128_g1 c, const uint32_t count, uint32_t *tmp_res, cgbn_mem_t<BITS>* tmp_buffer, cgbn_mem_t<BITS>* max_value, bool debug){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int instance = tid / TPI;
  if(instance >= count) return;

  context_t bn_context(cgbn_report_monitor, report, instance);
  env_t          bn_env(bn_context.env<env_t>());  

  DevAltBn128G1 dev_a, dev_b;
  dev_a.load(bn_env, a, instance);
  dev_b.load(bn_env, b, instance);

  if(dev_a.is_zero(bn_env)){
    dev_b.store(bn_env, c, instance);
    return;
  }
  if(dev_b.is_zero(bn_env)){
    dev_a.store(bn_env, c, instance);
    return;
  }

  const int n = BITS / 32;
  uint32_t *res = tmp_res + instance * 3 * n;
  cgbn_mem_t<BITS>* buffer = tmp_buffer + instance;
  env_t::cgbn_t tmax_value;
  cgbn_load(bn_env, tmax_value, max_value);


  //z1=squared(a.z)
  DevFp Z1 = dev_a.z.squared(bn_env, res, buffer);
  if(debug){
    Z1.print_64(bn_env, tmp_buffer);
  }
  //z2=squared(b.z)
  DevFp Z2 = dev_b.z.squared(bn_env, res, buffer);
  //u1=a.x * z2
  DevFp U1 = dev_a.x.mul(bn_env, Z2, res, buffer);
  //u2=b.x * z1
  DevFp U2 = dev_b.x.mul(bn_env, Z1, res, buffer);
  //z1_cubed = a.z * z1
  DevFp Z1_cubed = dev_a.z.mul(bn_env, Z1, res, buffer);
  //z2_cubed = b.z * z2
  DevFp Z2_cubed = dev_b.z.mul(bn_env, Z2, res, buffer);
  //s1 = a.y * z2_cubed
  DevFp S1 = dev_a.y.mul(bn_env, Z2_cubed, res, buffer);
  //s2 = b.y * z1_cubed
  DevFp S2 = dev_b.y.mul(bn_env, Z1_cubed, res, buffer);
  //if(u1 == u2) reutrn a.db1()
  if(U1.isequal(bn_env, U2) && S1.isequal(bn_env, S2)){
    if(threadIdx.x == 0){
      printf("dev dbl....\n");
    }
    dev_a.dbl(bn_env, c, tmp_res, tmp_buffer, tmax_value, instance);
    return;
  }

  //h = u2-u1
  DevFp H = U2.sub(bn_env, U1, tmax_value);
  //s2_minus_s1 = s2-s1
  DevFp S2_minus_S1 = S2.sub(bn_env, S1, tmax_value);
  //i = squared(h+h)
  DevFp h2 = H.add(bn_env, H, tmax_value);
  DevFp I = h2.squared(bn_env, res, buffer);
  //j = h * i
  DevFp J = H.mul(bn_env, I, res, buffer);
  //r = s2_minus_s1 + s2_minus_s1
  DevFp R = S2_minus_S1.add(bn_env, S2_minus_S1, tmax_value);
  //v = u1 * i
  DevFp V = U1.mul(bn_env, I, res, buffer);
  //x3 = square(r) - j - (v+v)
  DevFp r2 = R.squared(bn_env, res, buffer);
  DevFp v2 = V.add(bn_env, V, tmax_value);
  DevFp rj = r2.sub(bn_env, J, tmax_value);
  DevFp X3 = rj.sub(bn_env, v2, tmax_value);
  //s1_j = s1 * j
  DevFp S1_J = S1.mul(bn_env, J, res, buffer);
  //y3 = r * (v - x3) - (s1_j+s1_j)
  DevFp vx = V.sub(bn_env, X3, tmax_value);
  DevFp s1_j2 = S1_J.add(bn_env, S1_J, tmax_value);
  DevFp rvx = R.mul(bn_env, vx, res, buffer);
  DevFp Y3 = rvx.sub(bn_env, s1_j2, tmax_value);
  //z3 = (square(a.z + b.z) - z1 - z2) * h 
  DevFp abz = dev_a.z.add(bn_env, dev_b.z, tmax_value);
  DevFp abz2 = abz.squared(bn_env, res, buffer);
  DevFp abz2_z1 = abz2.sub(bn_env, Z1, tmax_value);
  DevFp abz2_z1_z2 = abz2_z1.sub(bn_env, Z2, tmax_value);
  DevFp Z3 = abz2_z1_z2.mul(bn_env, H, res, buffer);
  //c.x = x3 c.y = y3 c.z = z3
  dev_a.store(bn_env, X3, Y3, Z3, c, instance);
}

int alt_bn128_g1_add(alt_bn128_g1 a, alt_bn128_g1 b, alt_bn128_g1 c, const uint32_t count, uint32_t *tmp_res, cgbn_mem_t<BITS>* tmp_buffer, cgbn_mem_t<BITS>* max_value, bool debug){
  cgbn_error_report_t *report;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 

  uint32_t instances = std::min(count, (uint32_t)max_threads_per_block);
  uint32_t threads = instances * TPI;
  uint32_t blocks = (count + instances - 1) / instances;

  kernel_alt_bn128_g1_add<<<blocks, threads>>>(report, a, b, c, count, tmp_res, tmp_buffer, max_value, debug);

  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);
  CUDA_CHECK(cgbn_error_report_free(report));
  return 0;
}

}
