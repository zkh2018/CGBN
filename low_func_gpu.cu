
#include <gmp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cgbn/cgbn.h"
#include "low_func_gpu.h"
#include "low_func.cuh"
#include "gpu_support.h"

namespace gpu{

void gpu_mcl_add(uint32_t* z, uint32_t *x, uint32_t *y, uint32_t *p){
  //cgbn_error_report_t *report = nullptr;
  //CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  //kernel_mcl_add<<<1, 8>>>(report, z, x, y, p);
}

void gpu_mcl_sub(uint32_t* z, uint32_t *x, uint32_t *y, uint32_t *p){
  cgbn_error_report_t *report = nullptr;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  //kernel_mcl_sub<<<1, 8>>>(report, z, x, y, p);
}

void gpu_mcl_mul(uint32_t* z, uint32_t *x, uint32_t *y, uint32_t *p, const uint64_t rp, const bool print=false){
  cgbn_error_report_t *report = nullptr;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  if(print){
      //kernel_mcl_mul<true><<<1, 8>>>(report, z, x, y, p, rp);
  }else{
      //kernel_mcl_mul<false><<<1, 8>>>(report, z, x, y, p, rp);
  }
}

void gpu_mcl_ect_add(mcl_bn128_g1 R, mcl_bn128_g1 P, mcl_bn128_g1 Q, Fp_model one, Fp_model p, Fp_model a, const int specialA_, const int model_, const uint64_t rp){
  cgbn_error_report_t *report = nullptr;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  //kernel_ect_add<<<1, 8>>>(report, R, P, Q, one, p, a, specialA_, model_, rp);
  //kernel_ect_add_new<<<1, 1>>>(R, P, Q, one, p, a, specialA_, model_, rp);
}
void gpu_mcl_ect_add_new(mcl_bn128_g1 R, mcl_bn128_g1 P, mcl_bn128_g1 Q, Fp_model one, Fp_model p, Fp_model a, const int specialA_, const int model_, const uint64_t rp){
  kernel_ect_add_new<<<1, 1>>>(R, P, Q, one, p, a, specialA_, model_, rp);
}

void gpu_mcl_sub_g2(uint32_t* z, uint32_t *x, uint32_t *y, uint32_t *p){
  cgbn_error_report_t *report = nullptr;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  //kernel_mcl_sub_g2<<<1, 8>>>(report, z, x, y, p);
}
void gpu_mcl_add_g2(uint32_t* z, uint32_t *x, uint32_t *y, uint32_t *p){
  cgbn_error_report_t *report = nullptr;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  //kernel_mcl_add_g2<<<1, 8>>>(report, z, x, y, p);
}

void gpu_mont_red(uint32_t* z, uint32_t *xy, uint32_t *p, const uint64_t rp){
  cgbn_error_report_t *report = nullptr;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  //kernel_mont_red<<<1, 8>>>(report, z, xy, p, rp);
}

void gpu_mul_wide(uint32_t*z, uint32_t*x, uint32_t*y){
  cgbn_error_report_t *report = nullptr;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  //kernel_mul_wide<<<1, 8>>>(report, z, x, y);
}

void gpu_sub_wide(uint32_t*z, uint32_t*x, uint32_t*y){
  cgbn_error_report_t *report = nullptr;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  //kernel_sub_wide<<<1, 8>>>(report, z, x, y);
}
void gpu_fp2Dbl_mulPreW(uint32_t*z, uint32_t*x, uint32_t*y, uint32_t*p){
  cgbn_error_report_t *report = nullptr;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  //kernel_fp2Dbl_mulPreW<<<1, 8>>>(report, z, x, y, p);
}
void gpu_sqr_g2(uint32_t*y, uint32_t*x, uint32_t*p, const uint64_t rp){
  cgbn_error_report_t *report = nullptr;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  //kernel_sqr_g2<<<1, 8>>>(report, y, x, p, rp);
}
void gpu_mcl_mul_g2(uint32_t* z, uint32_t*x, uint32_t*y, uint32_t*p, const uint64_t rp){
  cgbn_error_report_t *report = nullptr;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  //kernel_mcl_mul_g2<<<1, 8>>>(report, z, x, y, p, rp);
}

void gpu_mcl_ect_add_g2(mcl_bn128_g2 R, mcl_bn128_g2 P, mcl_bn128_g2 Q, Fp_model one, Fp_model p, Fp_model2 a, const int specialA_, const int model_, const uint64_t rp){
  //cgbn_error_report_t *report = nullptr;
  //CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  cgbn_error_report_t *report = get_error_report();
  ///kernel_ect_add_g2<<<1, TPI>>>(report, R, P, Q, one, p, a, specialA_, model_, rp);
}

void gpu_mcl_ect_add_g2_new(mcl_bn128_g2 R, mcl_bn128_g2 P, mcl_bn128_g2 Q, Fp_model one, Fp_model p, Fp_model2 a, const int specialA_, const int model_, const uint64_t rp){
  //kernel_ect_add_g2_new<<<1, 1>>>(R, P, Q, one, p, a, specialA_, model_, rp);
}

} // namespace gpu

